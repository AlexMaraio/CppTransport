#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void backg( ulong n,
                                  double __Mp,
                                  $$__PARAM_ARGS,
                                  $$__COORD_ARGS{__x},
                                  $$__COORD_ARGS{__dxdt} )
  {
    size_t begin = blockDim.x * blockIdx.x + threadIdx.x;

    if(begin < n)
      {
        #define $$__COORDINATE[A] __x_$$__A[begin]

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        $$__TEMP_POOL{"double $1 = $2;"}

        for(size_t idx = begin, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
          {
            __dxdt_$$__A[idx] = $$__U1_PREDEF[A]{__Hsq, __eps};
          }
      }
  }
