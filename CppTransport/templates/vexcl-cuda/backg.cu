#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void backg( ulong n, $$__PARAM_ARGS, $$__COORD_ARGS{__x}, $$__COORD_ARGS{__dxdt}, double __Mp )
  {
    size_t begin = blockDim.x * blockIdx.x + threadIdx.x;

    if(begin < n)
      {
        #undef  $$__COORDINATE[A]
        #define $$__COORDINATE[A] __x_$$__A[begin]

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        #undef  $$__COORDINATE[A]
        #undef  __k
        #define $$__COORDINATE[A] __x_$$__A[idx]

        for(size_t idx = begin, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
          {
            $$__TEMP_POOL{"double $1 = $2;"}
            __dxdt_$$__A[idx] = $$__U1_PREDEF[A]{__Hsq, __eps};
          }
      }
  }
