#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

#if defined(_MSC_VER) || defined(__APPLE__)
typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;
#endif

extern "C" __global__ void threepffused( ulong __n,         // number of k-configurations we are integrating
                                         double __Mp,
                                         $$__PARAM_ARGS,
                                         double* __x,
                                         double* __dxdt,
                                         double* __k1_list,
                                         double* __k2_list,
                                         double* __k3_list,
                                         double __a )
  {
    size_t __begin = blockDim.x * blockIdx.x + threadIdx.x;

    #define __U2_SIZE      (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __U3_SIZE      (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __BACKG_SIZE   (2*$$__NUMBER_FIELDS)
    #define __TWOPF_SIZE   (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __THREEPF_SIZE (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)

    // packing of components into the state vector
    // (remember that all k-modes for a particular component are packed together, so reads can be coalesced)
    #define __BACKG_START       0
    #define __TWOPF_RE_K1_START __BACKG_START + __BACKG_SIZE
    #define __TWOPF_IM_K1_START __TWOPF_RE_K1_START + __TWOPF_SIZE
    #define __TWOPF_RE_K2_START __TWOPF_IM_K1_START + __TWOPF_SIZE
    #define __TWOPF_IM_K2_START __TWOPF_RE_K2_START + __TWOPF_SIZE
    #define __TWOPF_RE_K3_START __TWOPF_IM_K2_START + __TWOPF_SIZE
    #define __TWOPF_IM_K3_START __TWOPF_RE_K3_START + __TWOPF_SIZE
    #define __THREEPF_START     __TWOPF_IM_K3_START + __TWOPF_SIZE

    // convenience macros for accessing a particular components and k-configuration of the state vector
    #define __IN_BACKG(i,c)          __x[(__BACKG_START+i)*__n + c]
    #define __IN_TWOPF_RE_K1(i,j,c)  __x[(__TWOPF_RE_K1_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_TWOPF_IM_K1(i,j,c)  __x[(__TWOPF_IM_K1_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_TWOPF_RE_K2(i,j,c)  __x[(__TWOPF_RE_K2_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_TWOPF_IM_K2(i,j,c)  __x[(__TWOPF_IM_K2_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_TWOPF_RE_K3(i,j,c)  __x[(__TWOPF_RE_K3_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_TWOPF_IM_K3(i,j,c)  __x[(__TWOPF_IM_K3_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __IN_THREEPF(i,j,k,c)    __x[(__THREEPF_START+i*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS+j*2*$$__NUMBER_FIELDS+k)*__n + c]

    #define __OUT_BACKG(i,c)         __dxdt[(__BACKG_START+i)*__n + c]
    #define __OUT_TWOPF_RE_K1(i,j,c) __dxdt[(__TWOPF_RE_K1_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_TWOPF_IM_K1(i,j,c) __dxdt[(__TWOPF_IM_K1_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_TWOPF_RE_K2(i,j,c) __dxdt[(__TWOPF_RE_K2_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_TWOPF_IM_K2(i,j,c) __dxdt[(__TWOPF_IM_K2_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_TWOPF_RE_K3(i,j,c) __dxdt[(__TWOPF_RE_K3_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_TWOPF_IM_K3(i,j,c) __dxdt[(__TWOPF_IM_K3_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]
    #define __OUT_THREEPF(i,j,k,c)   __dxdt[(__THREEPF_START+i*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS+j*2*$$__NUMBER_FIELDS+k)*__n + c]

    // SPACE NEEDED IN SHARED MEMORY
    // k-modes: 3 per thread in the block   = 3*blockDim.x*sizeof(double)
    // u2-tensor: 3 per thread in the block = 3*__U2_SIZE*blockDim.x*sizeof(double)
    // u3-tensor: 3 per thread in the block = 3*__U3_SIZE*blockDim.x*sizeof(double)
    // twopf: 6 per thread in the block     = 6*__TWOPF_SIZE*blockDim.x*sizeof(double)
    // threepf: 1 per thread in the block   = __THREEPF_SIZE*blockDim.x*sizeof(double)

    // TOTAL                                = ( 3*(1 + __U2_SIZE + __U3_SIZE + 2*__TWOPF_SIZE) + __THREEPF_SIZE )*blockDim.x*sizeof(double)
    extern double __shared__ __smem[];

    // index into k-modes
    #define __k1 __smem[3*threadIdx.x]
    #define __k2 __smem[3*threadIdx.x+1]
    #define __k3 __smem[3*threadIdx.x+2]

    // index into u2-tensors
    #define __U2_k1(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __U2_k2(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 1*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __U2_k3(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 2*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    // index into u3-tensors
    #define __U3_k1k2k3(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]
    #define __U3_k2k1k3(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 1*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]
    #define __U3_k3k1k2(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 2*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]

    // index into twopfs
    #define __TWOPF_RE_k1(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k1(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 1*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_RE_k2(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 2*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k2(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 3*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_RE_k3(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 4*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k3(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 5*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    // index into threepf
    #define __THREEPF(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*blockDim.x*__TWOPF_SIZE + threadIdx.x*__THREEPF_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]

    if(__begin < __n)
      {
        size_t __grid_size = blockDim.x * gridDim.x;  // number of threads in the grid = stride size per kernel

        // read coords into local variables
        // we only need one, which we can choose to be '__begin'
        double $$__COORDINATE[A] = __IN_BACKG($$__A,__begin);

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        // cache u1 tensors for evolution of the background
        $$__TEMP_POOL{"double $1 = $2;"}
        double __u1_$$__A = $$__U1_PREDEF[A]{__Hsq, __eps};

        for(size_t __idx = __begin; __idx < __n; __idx += __grid_size)
          {
            // copy the data we need from global memory into shared memory
            __k1 = __k1_list[__idx];
            __k2 = __k2_list[__idx];
            __k3 = __k3_list[__idx];
            __TWOPF_RE_k1($$__A, $$__B)    = __IN_TWOPF_RE_K1($$__A,$$__B,__idx); $$//
            __TWOPF_IM_k1($$__A, $$__B)    = __IN_TWOPF_IM_K1($$__A,$$__B,__idx); $$//
            __TWOPF_RE_k2($$__A, $$__B)    = __IN_TWOPF_RE_K2($$__A,$$__B,__idx); $$//
            __TWOPF_IM_k2($$__A, $$__B)    = __IN_TWOPF_IM_K2($$__A,$$__B,__idx); $$//
            __TWOPF_RE_k3($$__A, $$__B)    = __IN_TWOPF_RE_K3($$__A,$$__B,__idx); $$//
            __TWOPF_IM_k3($$__A, $$__B)    = __IN_TWOPF_IM_K3($$__A,$$__B,__idx); $$//
            __THREEPF($$__A, $$__B, $$__C) = __IN_THREEPF($$__A,$$__B,$$__C,__idx); $$//

            $$__TEMP_POOL{"double $1 = $2;"}
            // compute u2 tensors
            __U2_k1($$__A, $$__B) = $$__U2_PREDEF[AB]{__k1, __a, __Hsq, __eps};
            __U2_k2($$__A, $$__B) = $$__U2_PREDEF[AB]{__k2, __a, __Hsq, __eps};
            __U2_k3($$__A, $$__B) = $$__U2_PREDEF[AB]{__k3, __a, __Hsq, __eps};

            // compute u3 tensors
            __U3_k1k2k3($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k1, __k2, __k3, __a, __Hsq, __eps};
            __U3_k2k1k3($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k2, __k1, __k3, __a, __Hsq, __eps};
            __U3_k3k1k2($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k3, __k1, __k2, __a, __Hsq, __eps};

            __OUT_BACKG($$__A,__idx) = __u1_$$__A; $$//

            __OUT_TWOPF_RE_K1($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__A,$$__C) * __TWOPF_RE_k1($$__C,$$__B);
            __OUT_TWOPF_RE_K1($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__B,$$__C) * __TWOPF_RE_k1($$__A,$$__C);

            __OUT_TWOPF_IM_K1($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__A,$$__C) * __TWOPF_IM_k1($$__C,$$__B);
            __OUT_TWOPF_IM_K1($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__B,$$__C) * __TWOPF_IM_k1($$__A,$$__C);

            __OUT_TWOPF_RE_K2($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__A,$$__C) * __TWOPF_RE_k2($$__C,$$__B);
            __OUT_TWOPF_RE_K2($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__B,$$__C) * __TWOPF_RE_k2($$__A,$$__C);

            __OUT_TWOPF_IM_K2($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__A,$$__C) * __TWOPF_IM_k2($$__C,$$__B);
            __OUT_TWOPF_IM_K2($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__B,$$__C) * __TWOPF_IM_k2($$__A,$$__C);

            __OUT_TWOPF_RE_K3($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__A,$$__C) * __TWOPF_RE_k3($$__C,$$__B);
            __OUT_TWOPF_RE_K3($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__B,$$__C) * __TWOPF_RE_k3($$__A,$$__C);

            __OUT_TWOPF_IM_K3($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__A,$$__C) * __TWOPF_IM_k3($$__C,$$__B);
            __OUT_TWOPF_IM_K3($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__B,$$__C) * __TWOPF_IM_k3($$__A,$$__C);

            __OUT_THREEPF($$__A,$$__B,$$__C,__idx)  = 0 $$// + $$__SUM_COORDS[M] __U2_k1($$__A,$$__M)*__THREEPF($$__M,$$__B,$$__C);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// + $$__SUM_2COORDS[MN] __U3_k1k2k3($$__A,$$__M,$$__N)*__TWOPF_RE_k2($$__M,$$__B)*__TWOPF_RE_k3($$__N,$$__C);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// - $$__SUM_2COORDS[MN] __U3_k1k2k3($$__A,$$__M,$$__N)*__TWOPF_IM_k2($$__M,$$__B)*__TWOPF_IM_k3($$__N,$$__C);

            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// + $$__SUM_COORDS[M] __U2_k2($$__B,$$__M)*__THREEPF($$__A,$$__M,$$__C);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// + $$__SUM_2COORDS[MN] __U3_k2k1k3($$__B,$$__M,$$__N)*__TWOPF_RE_k1($$__A,$$__M)*__TWOPF_RE_k3($$__N,$$__C);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// - $$__SUM_2COORDS[MN] __U3_k2k1k3($$__B,$$__M,$$__N)*__TWOPF_IM_k1($$__A,$$__M)*__TWOPF_IM_k3($$__N,$$__C);

            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// + $$__SUM_COORDS[M] __U2_k3($$__C,$$__M)*__THREEPF($$__A,$$__B,$$__M);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// + $$__SUM_2COORDS[MN] __U3_k3k1k2($$__C,$$__M,$$__N)*__TWOPF_RE_k1($$__A,$$__M)*__TWOPF_RE_k2($$__B,$$__N);
            __OUT_THREEPF($$__A,$$__B,$$__C,__idx) += 0 $$// - $$__SUM_2COORDS[MN] __U3_k3k1k2($$__C,$$__M,$$__N)*__TWOPF_IM_k1($$__A,$$__M)*__TWOPF_IM_k2($$__B,$$__N);
          }
      }
  }
