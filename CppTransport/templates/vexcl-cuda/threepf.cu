#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

#if defined(_MSC_VER) || defined(__APPLE__)
typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;
#endif

extern "C" __global__ void threepffused( ulong __n,
                                         double __Mp,
                                         $$__PARAM_ARGS,
                                         $$__COORD_ARGS{__x},
                                         double* __k1_list,
                                         double* __k2_list,
                                         double* __k3_list,
                                         double __a,
                                         $$__TWOPF_ARGS{__twopf_re_k1},
                                         $$__TWOPF_ARGS{__twopf_im_k1},
                                         $$__TWOPF_ARGS{__twopf_re_k2},
                                         $$__TWOPF_ARGS{__twopf_im_k2},
                                         $$__TWOPF_ARGS{__twopf_re_k3},
                                         $$__TWOPF_ARGS{__twopf_im_k3},
                                         $$__THREEPF_ARGS{__threepf},
                                         $$__COORD_ARGS{__dxdt},
                                         $$__TWOPF_ARGS{__dtwopf_re_k1},
                                         $$__TWOPF_ARGS{__dtwopf_im_k1},
                                         $$__TWOPF_ARGS{__dtwopf_re_k2},
                                         $$__TWOPF_ARGS{__dtwopf_im_k2},
                                         $$__TWOPF_ARGS{__dtwopf_re_k3},
                                         $$__TWOPF_ARGS{__dtwopf_im_k3},
                                         $$__THREEPF_ARGS{__dthreepf} )
  {
    size_t __begin = blockDim.x * blockIdx.x + threadIdx.x;

    #define __U2_SIZE      (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __U3_SIZE      (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __TWOPF_SIZE   (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __THREEPF_SIZE (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)

    // SPACE NEEDED IN SHARED MEMORY
    // k-modes: 3 per thread in the block   = 3*blockDim.x*sizeof(double)
    // u2-tensor: 3 per thread in the block = 3*__U2_SIZE*blockDim.x*sizeof(double)
    // u3-tensor: 3 per thread in the block = 3*__U3_SIZE*blockDim.x*sizeof(double)
    // twopf: 6 per thread in the block     = 6*__TWOPF_SIZE*blockDim.x*sizeof(double)
    // threepf: 1 per thread in the block   = __THREEPF_SIZE*blockDim.x*sizeof(double)

    // TOTAL                                = ( 3*(1 + __U2_SIZE + __U3_SIZE + 2*__TWOPF_SIZE) + __THREEPF_SIZE )*blockDim.x*sizeof(double)
    extern double __shared__ __smem[];

    // index into k-modes
    #define __k1 __smem[3*threadIdx.x]
    #define __k2 __smem[3*threadIdx.x+1]
    #define __k3 __smem[3*threadIdx.x+2]

    // index into u2-tensors
    #define __U2_k1(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __U2_k2(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 1*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __U2_k3(a,b) __smem[3*blockDim.x + 3*threadIdx.x*__U2_SIZE + 2*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    // index into u3-tensors
    #define __U3_k1k2k3(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]
    #define __U3_k2k1k3(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 1*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]
    #define __U3_k3k1k2(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*threadIdx.x*__U3_SIZE + 2*__U3_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]

    // index into twopfs
    #define __TWOPF_RE_k1(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k1(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 1*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_RE_k2(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 2*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k2(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 3*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_RE_k3(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 4*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]
    #define __TWOPF_IM_k3(a,b) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*threadIdx.x*__TWOPF_SIZE + 5*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    // index into threepf
    #define __THREEPF(a,b,c) __smem[3*blockDim.x + 3*blockDim.x*__U2_SIZE + 3*blockDim.x*__U3_SIZE + 6*blockDim.x*__TWOPF_SIZE + threadIdx.x*__THREEPF_SIZE + 2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS*a + 2*$$__NUMBER_FIELDS*b + c]

    if(__begin < __n)
      {
        size_t __grid_size = blockDim.x * gridDim.x;  // number of threads in the grid = stride size per kernel

        // read coords into local variables
        // we only need one, which we can choose to be '__begin'
        double $$__COORDINATE[A] = __x_$$__A[__begin];

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        // cache u1 tensors for evolution of the background
        $$__TEMP_POOL{"double $1 = $2;"}
        double __u1_$$__A = $$__U1_PREDEF[A]{__Hsq, __eps};

        for(size_t __idx = __begin; __idx < __n; __idx += __grid_size)
          {
            // copy the data we need from global memory into shared memory
            __k1 = __k1_list[__idx];
            __k2 = __k2_list[__idx];
            __k3 = __k3_list[__idx];
            __TWOPF_RE_k1($$__A, $$__B)    = __twopf_re_k1_$$__A_$$__B[__idx]; $$//
            __TWOPF_IM_k1($$__A, $$__B)    = __twopf_im_k1_$$__A_$$__B[__idx]; $$//
            __TWOPF_RE_k2($$__A, $$__B)    = __twopf_re_k2_$$__A_$$__B[__idx]; $$//
            __TWOPF_IM_k2($$__A, $$__B)    = __twopf_im_k2_$$__A_$$__B[__idx]; $$//
            __TWOPF_RE_k3($$__A, $$__B)    = __twopf_re_k3_$$__A_$$__B[__idx]; $$//
            __TWOPF_IM_k3($$__A, $$__B)    = __twopf_im_k3_$$__A_$$__B[__idx]; $$//
            __THREEPF($$__A, $$__B, $$__C) = __threepf_$$__A_$$__B_$$__C[__idx]; $$//

            $$__TEMP_POOL{"double $1 = $2;"}
            // compute u2 tensors
            __U2_k1($$__A, $$__B) = $$__U2_PREDEF[AB]{__k1, __a, __Hsq, __eps};
            __U2_k2($$__A, $$__B) = $$__U2_PREDEF[AB]{__k2, __a, __Hsq, __eps};
            __U2_k3($$__A, $$__B) = $$__U2_PREDEF[AB]{__k3, __a, __Hsq, __eps};

            // compute u3 tensors
            __U3_k1k2k3($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k1, __k2, __k3, __a, __Hsq, __eps};
            __U3_k2k1k3($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k2, __k1, __k3, __a, __Hsq, __eps};
            __U3_k3k1k2($$__A, $$__B, $$__C) = $$__U3_PREDEF[ABC]{__k3, __k1, __k2, __a, __Hsq, __eps};

            __dxdt_$$__A[__idx] = __u1_$$__A; $$//

            double __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__A,$$__C) * __TWOPF_RE_k1($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__B,$$__C) * __TWOPF_RE_k1($$__A,$$__C);
            __dtwopf_re_k1_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__A,$$__C) * __TWOPF_IM_k1($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k1($$__B,$$__C) * __TWOPF_IM_k1($$__A,$$__C);
            __dtwopf_im_k1_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__A,$$__C) * __TWOPF_RE_k2($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__B,$$__C) * __TWOPF_RE_k2($$__A,$$__C);
            __dtwopf_re_k2_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__A,$$__C) * __TWOPF_IM_k2($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k2($$__B,$$__C) * __TWOPF_IM_k2($$__A,$$__C);
            __dtwopf_im_k2_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__A,$$__C) * __TWOPF_RE_k3($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__B,$$__C) * __TWOPF_RE_k3($$__A,$$__C);
            __dtwopf_re_k3_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            __cache_twopf_$$__A_$$__B  = 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__A,$$__C) * __TWOPF_IM_k3($$__C,$$__B);
            __cache_twopf_$$__A_$$__B += 0 $$// + $$__SUM_COORDS[C] __U2_k3($$__B,$$__C) * __TWOPF_IM_k3($$__A,$$__C);
            __dtwopf_im_k3_$$__A_$$__B[__idx] = __cache_twopf_$$__A_$$__B; $$//

            double __cache_threepf_$$__A_$$__B_$$__C; $$//

            __cache_threepf_$$__A_$$__B_$$__C  = 0 $$// + $$__SUM_COORDS[M] __U2_k1($$__A,$$__M)*__THREEPF($$__M,$$__B,$$__C);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// + $$__SUM_2COORDS[MN] __U3_k1k2k3($$__A,$$__M,$$__N)*__TWOPF_RE_k2($$__M,$$__B)*__TWOPF_RE_k3($$__N,$$__C);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// - $$__SUM_2COORDS[MN] __U3_k1k2k3($$__A,$$__M,$$__N)*__TWOPF_IM_k2($$__M,$$__B)*__TWOPF_IM_k3($$__N,$$__C);

            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// + $$__SUM_COORDS[M] __U2_k2($$__B,$$__M)*__THREEPF($$__A,$$__M,$$__C);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// + $$__SUM_2COORDS[MN] __U3_k2k1k3($$__B,$$__M,$$__N)*__TWOPF_RE_k1($$__A,$$__M)*__TWOPF_RE_k3($$__N,$$__C);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// - $$__SUM_2COORDS[MN] __U3_k2k1k3($$__B,$$__M,$$__N)*__TWOPF_IM_k1($$__A,$$__M)*__TWOPF_IM_k3($$__N,$$__C);

            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// + $$__SUM_COORDS[M] __U2_k3($$__C,$$__M)*__THREEPF($$__A,$$__B,$$__M);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// + $$__SUM_2COORDS[MN] __U3_k3k1k2($$__C,$$__M,$$__N)*__TWOPF_RE_k1($$__A,$$__M)*__TWOPF_RE_k2($$__B,$$__N);
            __cache_threepf_$$__A_$$__B_$$__C += 0 $$// - $$__SUM_2COORDS[MN] __U3_k3k1k2($$__C,$$__M,$$__N)*__TWOPF_IM_k1($$__A,$$__M)*__TWOPF_IM_k2($$__B,$$__N);
            __dthreepf_$$__A_$$__B_$$__C[__idx] = __cache_threepf_$$__A_$$__B_$$__C; $$//
          }
      }
  }
