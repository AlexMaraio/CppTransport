#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void twopffused( ulong n,
  $$__TWOPF_ARGS{__twopf_re_k1},
  $$__TWOPF_ARGS{__twopf_im_k1},
  $$__TWOPF_ARGS{__twopf_re_k2},
  $$__TWOPF_ARGS{__twopf_im_k2},
  $$__TWOPF_ARGS{__twopf_re_k3},
  $$__TWOPF_ARGS{__twopf_im_k3},
  $$__THREEPF_ARGS{__threepf},
  $$__THREEPF_ARGS{__dthreepf},
  $$__U2_ARGS{__u2_k1},
  $$__U2_ARGS{__u2_k2},
  $$__U2_ARGS{__u2_k3},
  $$__U3_ARGS{__u3_k1k2k3},
  $$__U3_ARGS{__u3_k2k1k3},
  $$__U3_ARGS{__u3_k3k1k2} )
  {
    for(size_t idx = blockDim.x * blockIdx.x + threadIdx.x, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
      {
        __dthreepf_$$__A_$$__B_$$__C[idx]  = 0 $$// + $$__U2_NAME[AM]{__u2_k1}[idx]*__threepf_$$__M_$$__B_$$__C[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// + $$__U3_NAME[AMN]{__u3_k1k2k3}[idx]*__twopf_re_k2_$$__M_$$__B[idx]*__twopf_re_k3_$$__N_$$__C[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// - $$__U3_NAME[AMN]{__u3_k1k2k3}[idx]*__twopf_im_k2_$$__M_$$__B[idx]*__twopf_im_k3_$$__N_$$__C[idx];

        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// + $$__U2_NAME[BM]{__u2_k2}[idx]*__threepf_$$__A_$$__M_$$__C[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// + $$__U3_NAME[BMN]{__u3_k2k1k3}[idx]*__twopf_re_k1_$$__A_$$__M[idx]*__twopf_re_k3_$$__N_$$__C[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// - $$__U3_NAME[BMN]{__u3_k2k1k3}[idx]*__twopf_im_k1_$$__A_$$__M[idx]*__twopf_im_k3_$$__N_$$__C[idx];

        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// + $$__U2_NAME[CM]{__u2_k3}[idx]*__threepf_$$__A_$$__B_$$__M[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// + $$__U3_NAME[CMN]{__u3_k3k1k2}[idx]*__twopf_re_k1_$$__A_$$__M[idx]*__twopf_re_k2_$$__B_$$__N[idx];
        __dthreepf_$$__A_$$__B_$$__C[idx] += 0 $$// - $$__U3_NAME[CMN]{__u3_k3k1k2}[idx]*__twopf_im_k1_$$__A_$$__M[idx]*__twopf_im_k2_$$__B_$$__N[idx];
      }
  }
