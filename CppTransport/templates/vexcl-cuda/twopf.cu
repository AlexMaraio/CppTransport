#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

#if defined(_MSC_VER) || defined(__APPLE__)
typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;
#endif

extern "C" __global__ void twopffused( ulong __n,         // number of k-configurations we are integrating
                                       double __Mp,
                                       $$__PARAM_ARGS,
                                       double* __x,
                                       double* __dxdt,
                                       double* __klist,
                                       double __a )
  {
    size_t __begin = blockDim.x * blockIdx.x + threadIdx.x;

    #define __U2_SIZE    (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)
    #define __BACKG_SIZE (2*$$__NUMBER_FIELDS)
    #define __TWOPF_SIZE (2*$$__NUMBER_FIELDS*2*$$__NUMBER_FIELDS)

    // packing of components into the state vector
    // (remember that all k-modes for a particular component are packed together, so reads can be coalesced)
    #define __BACKG_START 0
    #define __TWOPF_START __BACKG_START + __BACKG_SIZE

    // convenience macros for accessing a particular components and k-configuration of the state vector
    #define __IN_BACKG(i,c)    __x[(__BACKG_START+i)*__n + c]
    #define __IN_TWOPF(i,j,c)  __x[(__TWOPF_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]

    #define __OUT_BACKG(i,c)   __dxdt[(__BACKG_START+i)*__n + c]
    #define __OUT_TWOPF(i,j,c) __dxdt[(__TWOPF_START+i*2*$$__NUMBER_FIELDS+j)*__n + c]

    // SPACE NEEDED IN SHARED MEMORY:
    // k-modes: 1 per thread in the block   = sizeof(double)*blockDim.x
    //                                      = 8*blockDim.x bytes
    // u2-tensor: 1 per thread in the block = __U2_SIZE*blockDim.x*sizeof(double)
    //                                      = 16*8*blockDim.x bytes = 128*blockDim.x bytes
    // twopf: 1 per thread in the block     = __TWOPF_SIZE*blockDim.x*sizeof(double)
    //                                      = 16*8*blockDim.x bytes = 128*blockDim.x bytes

    // TOTAL                                = (8+128+128)*blockDim.x bytes = 264*blockDim.x bytes
    extern double __shared__ __smem[];

    // index into k-modes
    #define __k __smem[threadIdx.x]

    // index into u2 tensor
    // we allocate all u2 tensors in a block after the k-modes, so have to offset by blockDim.x
    #define __U2(a,b) __smem[blockDim.x + threadIdx.x*__U2_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    // index into twopf
    // twopfs come in a block after the u2-tensors, so we have to offset by blockDim.x + __U2_SIZE*blockDim.x
    #define __TWOPF(a,b) __smem[blockDim.x + __U2_SIZE*blockDim.x + threadIdx.x*__TWOPF_SIZE + 2*$$__NUMBER_FIELDS*a + b]

    if(__begin < __n)
      {
        size_t __grid_size = blockDim.x * gridDim.x;  // number of threads in the grid = stride size per kernel

        // read coords into local variables
        // we only need one, which we can choose to be '__begin'
        double $$__COORDINATE[A] = __IN_BACKG($$__A,__begin);

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        // cache u1 tensors for evolution of the background
        $$__TEMP_POOL{"double $1 = $2;"}
        double __u1_$$__A = $$__U1_PREDEF[A]{__Hsq, __eps};

        for(size_t __idx = __begin; __idx < __n; __idx += __grid_size)
          {
            // copy the data we need from global memory into shared memory
            __k = __klist[__idx];
            __TWOPF($$__A, $$__B) = __IN_TWOPF($$__A,$$__B,__idx); $$//

            // compute u2 for the k-mode we are looking at
            $$__TEMP_POOL{"double $1 = $2;"}

            __U2($$__A,$$__B) = $$__U2_PREDEF[AB]{__k, __a, __Hsq, __eps};

            __OUT_BACKG($$__A,__idx) = __u1_$$__A; $$//

            __OUT_TWOPF($$__A,$$__B,__idx)  = 0 $$// + $$__SUM_COORDS[C] __U2($$__A,$$__C) * __TWOPF($$__C,$$__B);
            __OUT_TWOPF($$__A,$$__B,__idx) += 0 $$// + $$__SUM_COORDS[C] __U2($$__B,$$__C) * __TWOPF($$__A,$$__C);
          }
      }
  }
