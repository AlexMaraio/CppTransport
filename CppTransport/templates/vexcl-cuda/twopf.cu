#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void twopffused( ulong n, $$__TWOPF_ARGS{__twopf}, $$__TWOPF_ARGS{__dtwopf}, $$__U2_ARGS{__u2} )
  {
    for(size_t idx = blockDim.x * blockIdx.x + threadIdx.x, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
      {
        __dtwopf_$$__A_$$__B[idx]  = 0 $$// + $$__U2_NAME[AC]{__u2}[idx] * __twopf_$$__C_$$__B[idx];
        __dtwopf_$$__A_$$__B[idx] += 0 $$// + $$__U2_NAME[BC]{__u2}[idx] * __twopf_$$__A_$$__C[idx];
      }
  }
