#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void u2fused( ulong n, $$__PARAM_ARGS, $$__COORD_ARGS{__x}, $$__U2_ARGS{__u2}, double* __klist, double __a, double __Mp )
  {
    size_t begin = blockDim.x * blockIdx.x + threadIdx.x;

    if(begin < n)
      {
        #undef  $$__COORDINATE[A]
        #define $$__COORDINATE[A] __x_$$__A[begin]

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        #undef  $$__COORDINATE[A]
        #undef  __k
        #define $$__COORDINATE[A] __x_$$__A[idx]
        #define __k __klist[idx]

        for(size_t idx = begin, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
          {
            $$__TEMP_POOL{"double $1 = $2;"}
            __u2_$$__A_$$__B[idx] = $$__U2_PREDEF[AB]{__k, __a, __Hsq, __eps};
          }
      }
  }
