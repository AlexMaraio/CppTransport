#include "hip/hip_runtime.h"
// backend=cuda, minver=0.06

typedef unsigned char       uchar;
typedef unsigned int        uint;
typedef unsigned short      ushort;
typedef unsigned long long  ulong;

extern "C" __global__ void u3fused( ulong n, $$__PARAM_ARGS, $$__COORD_ARGS{__x}, $$__U3_ARGS{__u3}, double* __k1list, double* __k2list, double* __k3list, double __a, double __Mp )
  {
    size_t begin = blockDim.x * blockIdx.x + threadIdx.x;

    if(begin < n)
    {
        #undef  $$__COORDINATE[A]
        #define $$__COORDINATE[A] __x_$$__A[begin]

        double __Hsq = $$__HUBBLE_SQ;
        double __eps = $$__EPSILON;

        #undef  $$__COORDINATE[A]
        #undef  __k1
        #undef  __k2
        #undef  __k3
        #define $$__COORDINATE[A] __x_$$__A[idx]
        #define __k1 __k1list[idx]
        #define __k2 __k2list[idx]
        #define __k3 __k3list[idx]

        for(size_t idx = begin, grid_size = blockDim.x * gridDim.x; idx < n; idx += grid_size)
          {
            $$__TEMP_POOL{"double $1 = $2;"}
            __u3_$$__A_$$__B_$$__C[idx] = $$__U3_PREDEF[ABC]{__k1, __k2, __k3, __a, __Hsq, __eps};
          }
      }
  }
